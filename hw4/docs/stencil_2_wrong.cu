#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 128
void check(int N, const float *A, float *x, float *y, float *y_target, float bc_initial, float bc_final) {
    // Perform matrix-vector multiplication
    for (int i = 0; i < N; ++i) {
        float sum = 0.0f;  // Initialize the sum for the i-th element of y
        for (int j = 0; j < N; ++j) {
            sum += A[j*N+i] * x[j];  // Accumulate the dot product of the i-th row of A and x
        }
        y_target[i] = sum;  // Store the result in y_target
    }
    y_target[0] = bc_initial;
    y_target[N-1] = bc_final;

    // Check element-wise for accuracy
    bool isCorrect = true;
    float tol = 1e-9;
    for (int i = 0; i < N; ++i) {
        float diff = fabs(y[i] - y_target[i]);  // Calculate the difference between computed and target
        if (diff >= tol) {  // If the difference exceeds the tolerance, the result is incorrect
            printf("y is not accurate to machine precision.\n");
            printf("At index %d, incorrect element = %f, correct element = %f\n", i, y[i], y_target[i]);
            isCorrect = false;
            break;  // Exit early on the first error
        }
    }
    if (isCorrect) {
        printf("y is accurate to machine precision.\n");
    }
}


__global__ void stencil_global(int N, const float *A, float *x, float *y, float bc_initial, float bc_final){
  __shared__ float s_x[BLOCKSIZE];
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  const int tid = threadIdx.x;
  
  // Write in A
  

  // Write in x
  s_x[tid] = 0.f
  if(x<N){
    s_x[tid] = x[i];
  }

  // Compute y from shared memory
  if (i < N){
    float val = y[i];
    for (int j = 0; j < N; ++j){
      val += A[j*N+i] * x[j]; 
    }
    y[i] = val;
  }

  if(i==0) y[i] = bc_initial;
  else if(i==N-1) y[i] = bc_final;

  
}
    
int main(int argc, char * argv[]){
  int N = 4096;
  if (argc > 1){
    N = atoi(argv[1]);
  }

  int blockSize = BLOCKSIZE;

  // Next largest multiple of blockSize
  int numBlocks = (N + blockSize - 1) / blockSize;

  printf("N = %d, blockSize = %d, numBlocks = %d\n", N, blockSize, numBlocks);
  
  
  float * A = new float[blockSize*blockSize]; // fit the block sizes
  float * x = new float[N];
  float * y = new float[N];
  float * y_target = new float[N];
  for (int i = 0; i < N; ++i) {
    // Initialize x and y
    x[i] = 1.f;
    y[i] = 0.f;    
    y_target[i] = 0.f;
    // Set indices for A (symmetric)
    int main = i*N+i;
    int super = i*N+(i+1);
    int sub = i*N+(i-1);
    // Set elements
    A[main] = 2.f;
    if (i < N - 1) {
        A[super] = -1.f; 
    }
    if (i > 0) {
        A[sub] = -1.f; 
    }
  } 

  // Define boundary conditions
  float bc_initial = 50.f;
  float bc_final = 50.f;

  // allocate memory and copy to the GPU
  float * d_A;
  float * d_x;
  float * d_y;
  int size_A = N*N*sizeof(float);  
  int size_x = N*sizeof(float);
  int size_y = N*sizeof(float);
  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_y, size_y);
  hipMalloc((void **) &d_A, size_A);
  
  // copy memory over to the GPU
  hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size_y, hipMemcpyHostToDevice);
  stencil_global <<< numBlocks, blockSize >>> (N, d_A, d_x, d_y, bc_initial, bc_final);

  // copy memory back to the CPU
  hipMemcpy(y, d_y, size_y, hipMemcpyDeviceToHost);
  
  // Compute target for stencil and check for accuracy
  check(N, A, x, y, y_target, bc_initial, bc_final);

#if 1
  int num_trials = 10;
  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < num_trials; ++i){
    stencil_global <<< numBlocks, blockSize >>> (N, d_A, d_x, d_y, bc_initial, bc_final);
  }


  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  
  printf("Time to run kernel 10x: %6.2f ms.\n", time);
  
#endif

  return 0;
}