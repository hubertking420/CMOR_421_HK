#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 128
__global__ void partial_reduction(const int N, float *x_reduced, const float *x){
    __shared__ float s_x[BLOCKSIZE];
    const int tid = threadIdx.x;
    const int i = blockDim.x * blockIdx.x + tid; // Index in the first half
    const int j = N - 1 - tid - blockDim.x * blockIdx.x; // Index in the second half

    // Load data into shared memory, ensuring we don't read out of bounds
    s_x[tid] = (i < N) ? x[i] : 0.f;
    if (j >= 0 && j < N) {
        s_x[tid] += x[j];
    }

    __syncthreads(); // Ensure all threads have written their sums to shared memory

    // Perform the reduction in shared memory
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            s_x[index] += s_x[index + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0){
        x_reduced[blockIdx.x] = s_x[0];
    }
}   
int main(int argc, char * argv[]){
    int N = 419430;
    if (argc > 1){
        N = atoi(argv[1]);
    }

    int blockSize = BLOCKSIZE;

    // Next largest multiple of blockSize
    int numBlocks = (N + blockSize - 1) / blockSize;
    numBlocks /= 2;
    printf("Reduction with N = %d, blockSize = %d, numBlocks = %d\n", N, blockSize, numBlocks);

    float * x = new float[N];
    float * x_reduced = new float[numBlocks];  

    for (int i = 0; i < N; ++i){
        x[i] = i + 1.f;
    }

    // allocate memory and copy to the GPU
    float * d_x;
    float * d_x_reduced;  
    int size_x = N * sizeof(float);
    int size_x_reduced = numBlocks * sizeof(float);
    hipMalloc((void **) &d_x, size_x);
    hipMalloc((void **) &d_x_reduced, size_x_reduced);
  
    // copy memory over to the GPU
    hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
    hipMemcpy(d_x_reduced, x_reduced, size_x_reduced, hipMemcpyHostToDevice);

    partial_reduction <<< numBlocks, blockSize >>> (N, d_x_reduced, d_x);
    hipError_t code = hipGetLastError();
    if (code != hipSuccess){
        printf("GPUassert: %s\n", hipGetErrorString(code));
    }
    // copy memory back to the CPU
    hipMemcpy(x_reduced, d_x_reduced, size_x_reduced, hipMemcpyDeviceToHost);

    float sum_x = 0.f;
    for (int i = 0; i < numBlocks; ++i){
        sum_x += x_reduced[i];
    }

    float target = N * (N+1) / 2.f;
    printf("error = %f\n", fabs(sum_x - target));

#if 1
    int num_trials = 10;
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < num_trials; ++i){
        partial_reduction <<< numBlocks, blockSize >>> (N, d_x_reduced, d_x);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
  
    printf("Time to run kernel 10x: %6.2f ms.\n", time);
#endif

    return 0;
}